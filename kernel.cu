#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void VecAdd(const float* A, const float* B, float* C, int N) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < N;
         i += blockDim.x * gridDim.x) {
        C[i] = A[i] + B[i];
         }
}
