#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "kernel.cu"
#include <chrono>
#include <fstream>

int main (int argc, char *argv[]){

    float *A_h, *B_h, *C_h;
    float *A_d, *B_d, *C_d;
    unsigned VecSize;

    if (argc == 1) {
        VecSize = 256;
    } else if (argc == 2) {
      VecSize = atoi(argv[1]);
    } else {
        printf("Usage: ./vecAdd <Size>");
        exit(0);
    }

    A_h = (float*) malloc( sizeof(float) * VecSize );
    B_h = (float*) malloc( sizeof(float) * VecSize );
    C_h = (float*) malloc( sizeof(float) * VecSize );

      for (unsigned int i=0; i < VecSize; i++) {
      A_h[i] = i;
      B_h[i] = i;
    }

    hipDeviceSynchronize();

    int size = VecSize * sizeof(float); float *d_A, *d_B, *d_C;
    hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &d_B, size);
    hipMalloc((void **) &d_C, size);
    hipMemcpy(d_A, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B_h, size, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    // int threadsPerBlock = 256;
    int n = (int)VecSize;
    // int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    // VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

    // hipMemcpy(C_h, d_C, size, hipMemcpyDeviceToHost);


    // hipDeviceSynchronize();

    // for (int i = 0; i < 8 && i < n; ++i) {
    //     printf("C[%d] = %f\n", i, C_h[i]);
    // }

    int threadsPerBlock[] = {1,2,4,8,16,32,64,128,256,512,1024};
    int Blocks[]      = {1,2,4,8,16,32,64,128,256,512,1024};
    int THREAD_SIZE = sizeof(threadsPerBlock)/sizeof(threadsPerBlock[0]);
    int BLOCK_SIZE  = sizeof(Blocks)/sizeof(Blocks[0]);




  hipDeviceProp_t prop{};
  hipGetDeviceProperties(&prop, 0);

  std::ofstream csv("timings.csv");
  csv << "blocks,threads,n,time_ms\n";

  for (int bi = 0; bi < BLOCK_SIZE; ++bi) {
      int B = Blocks[bi];


      for (int ti = 0; ti < THREAD_SIZE; ++ti) {
          int T = threadsPerBlock[ti];



          printf("B=%d, T=%d :\n", B, T);
          auto t0 = std::chrono::high_resolution_clock::now();
          VecAdd<<<B, T>>>(d_A, d_B, d_C, n);


          hipError_t err = hipGetLastError();
          if (err != hipSuccess) {
              fprintf(stderr, "Launch failed (B=%d,T=%d): %s\n",
                      B, T, hipGetErrorString(err));
              continue;
          }

          hipDeviceSynchronize();
          auto t1 = std::chrono::high_resolution_clock::now();
          double ms = std::chrono::duration<double, std::milli>(t1 - t0).count();

          hipMemcpy(C_h, d_C, size, hipMemcpyDeviceToHost);

          csv << B << ',' << T << ',' << n << ',' << ms << '\n';

          printf("B=%4d T=%4d | C[0]=%.1f C[1]=%.1f C[%d]=%.1f\n",
                B, T, C_h[0], C_h[1], n-1, C_h[n-1]);
      }
  }

  csv.close();



    free(A_h);
    free(B_h);
    free(C_h);

    //INSERT Memory CODE HERE
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}